#include "hip/hip_runtime.h"
#include "motion_compensation.h"

#include <cmath>
#include <algorithm> //for std::max
#include <cstdio>
#include <vector>
#include <iostream>
#include "utils.h"

#include <hipcub/hipcub.hpp>
#include <cub/block/block_reduce.cuh>

#include <jetson-utils/cudaMappedMemory.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define FULL_MASK 0xffffffff
// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template <class T>
struct SharedMemory
{
    __device__ inline operator T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

__global__ void fillImageBilinear_(float fx, float fy, float cx, float cy, int height, int width, int num_events, const float *x_unprojected, const float *y_unprojected, float *x_prime, float *y_prime, float *t, float *image, const float rotation_x, const float rotation_y, const float rotation_z, float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum)
{

    float image_sum = 0;
    float image_sum_del_theta_x = 0;
    float image_sum_del_theta_y = 0;
    float image_sum_del_theta_z = 0;
    float *image_del_x = image + height * width;
    float *image_del_y = image + height * width * 2;
    float *image_del_z = image + height * width * 3;
    size_t i = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    // size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);
    if (i < num_events)
    {
        // calculate theta x,y,z
        float theta_x_t = rotation_x * t[i];
        float theta_y_t = rotation_y * t[i];
        float theta_z_t = rotation_z * t[i];

        // calculate x/y/z_rotated
        float z_rotated_inv = 1 / (-theta_y_t * x_unprojected[i] + theta_x_t * y_unprojected[i] + 1);
        float x_rotated_norm = (x_unprojected[i] - theta_z_t * y_unprojected[i] + theta_y_t) * z_rotated_inv;
        float y_rotated_norm = (theta_z_t * x_unprojected[i] + y_unprojected[i] - theta_x_t) * z_rotated_inv;

        // calculate x_prime and y_prime
        x_prime[i] = fx * x_rotated_norm + cx;
        y_prime[i] = fy * y_rotated_norm + cy;
        // populate image

        // Bilinear
        int x_trunc = int(x_prime[i]);
        int y_trunc = int(y_prime[i]);
        if (x_trunc >= 1 && x_trunc <= width - 2 && y_trunc >= 1 && y_trunc <= height - 2)
        {

            // int idx1 = x_trunc - 1 + (y_trunc - 1) * width;
            // int idx2 = idx1 + 1;
            // int idx3 = idx1 + width;
            // int idx4 = idx3 + 1;

            int idx4 = x_trunc + y_trunc * width;
            int idx3 = idx4 - 1;
            int idx2 = idx4 - width;
            int idx1 = idx2 - 1;
            float x_diff = x_prime[i] - x_trunc;
            float y_diff = y_prime[i] - y_trunc;
            float del_x_del_theta_x, del_x_del_theta_y, del_x_del_theta_z, del_y_del_theta_x, del_y_del_theta_y, del_y_del_theta_z;
            float fx_div_z_rotated_ti = fx * z_rotated_inv * t[i];
            float fy_div_z_rotated_ti = fy * z_rotated_inv * t[i];
            del_x_del_theta_y = fx_div_z_rotated_ti * (1 + x_unprojected[i] * x_rotated_norm);
            del_x_del_theta_z = fx_div_z_rotated_ti * -y_unprojected[i];
            del_x_del_theta_x = del_x_del_theta_z * x_rotated_norm;
            del_y_del_theta_x = fy_div_z_rotated_ti * (-1 - y_unprojected[i] * y_rotated_norm);
            del_y_del_theta_z = fy_div_z_rotated_ti * x_unprojected[i];
            del_y_del_theta_y = del_y_del_theta_z * y_rotated_norm;
            // float d1x = -(1 - y_diff);
            // float d1y = -(1 - x_diff);
            float d2x = 1 - y_diff;
            float d2y = -x_diff;
            float d3x = -y_diff;
            float d3y = 1 - x_diff;
            float d4x = y_diff;
            float d4y = x_diff;

            float d1x = -d2x;
            float d1y = -d3y;

            // float im1 = (1 - x_diff) * (1 - y_diff);
            float im1 = d3y * d2x;
            // float im2 = (x_diff) * (1 - y_diff);
            float im2 = d4y * d2x;
            // float im3 = (1 - x_diff) * (y_diff);
            float im3 = d3y * y_diff;
            float im4 = (x_diff) * (y_diff);
            image_sum = im1 + im2 + im3 + im4;
            atomicAdd(&image[idx1], im1);
            atomicAdd(&image[idx2], im2);
            atomicAdd(&image[idx3], im3);
            atomicAdd(&image[idx4], im4);
            float dx1 = d1x * del_x_del_theta_x + d1y * del_y_del_theta_x;
            float dx2 = d2x * del_x_del_theta_x + d2y * del_y_del_theta_x;
            float dx3 = d3x * del_x_del_theta_x + d3y * del_y_del_theta_x;
            float dx4 = d4x * del_x_del_theta_x + d4y * del_y_del_theta_x;
            image_sum_del_theta_x = dx1 + dx2 + dx3 + dx4;

            atomicAdd(&image_del_x[idx1], dx1);
            atomicAdd(&image_del_x[idx2], dx2);
            atomicAdd(&image_del_x[idx3], dx3);
            atomicAdd(&image_del_x[idx4], dx4);
            float dy1 = d1x * del_x_del_theta_y + d1y * del_y_del_theta_y;
            float dy2 = d2x * del_x_del_theta_y + d2y * del_y_del_theta_y;
            float dy3 = d3x * del_x_del_theta_y + d3y * del_y_del_theta_y;
            float dy4 = d4x * del_x_del_theta_y + d4y * del_y_del_theta_y;
            image_sum_del_theta_y = dy1 + dy2 + dy3 + dy4;
            atomicAdd(&image_del_y[idx1], dy1);
            atomicAdd(&image_del_y[idx2], dy2);
            atomicAdd(&image_del_y[idx3], dy3);
            atomicAdd(&image_del_y[idx4], dy4);
            float dz1 = d1x * del_x_del_theta_z + d1y * del_y_del_theta_z;
            float dz2 = d2x * del_x_del_theta_z + d2y * del_y_del_theta_z;
            float dz3 = d3x * del_x_del_theta_z + d3y * del_y_del_theta_z;
            float dz4 = d4x * del_x_del_theta_z + d4y * del_y_del_theta_z;
            image_sum_del_theta_z = dz1 + dz2 + dz3 + dz4;
            atomicAdd(&image_del_z[idx1], dz1);
            atomicAdd(&image_del_z[idx2], dz2);
            atomicAdd(&image_del_z[idx3], dz3);
            atomicAdd(&image_del_z[idx4], dz4);
        }
    }
    float *sdata = SharedMemory<float>();
    uint16_t tid = threadIdx.x;

    // do reduction in shared mem

    // sum up to 128 elements

    float temp_sum;
    // image_sum
    sdata[tid] = image_sum;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_sum = image_sum + sdata[tid + 256];
    __syncthreads();
    // store contrast in 0 to 127
    if (tid < 128)
        temp_sum = image_sum + sdata[tid + 128];
    __syncthreads();
    // image_sum_del_theta_x
    sdata[tid] = image_sum_del_theta_x;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_sum_del_theta_x = image_sum_del_theta_x + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_sum_del_theta_x = image_sum_del_theta_x + sdata[tid + 128];
    __syncthreads();
    // store x in 128 to 255
    if (tid >= 128 && tid < 256)
    {
        temp_sum = sdata[tid - 128];
    }
    __syncthreads();
    // image_sum_del_theta_y
    sdata[tid] = image_sum_del_theta_y;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_sum_del_theta_y = image_sum_del_theta_y + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_sum_del_theta_y = image_sum_del_theta_y + sdata[tid + 128];
    __syncthreads();
    // store y in 256 to 383
    if (tid >= 256 && tid < 384)
    {
        temp_sum = sdata[tid - 256];
    }
    __syncthreads();
    // image_sum_del_theta_z
    sdata[tid] = image_sum_del_theta_z;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_sum_del_theta_z = image_sum_del_theta_z + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
    {
        sdata[tid] = image_sum_del_theta_z = image_sum_del_theta_z + sdata[tid + 128];
    }
    __syncthreads();
    // store z in 384 to 512
    if (tid >= 384)
    {
        temp_sum = sdata[tid - 384];
    }
    // dump partial sums inside again
    sdata[tid] = temp_sum;
    __syncthreads();
    if ((tid & 0x7F) < 64)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 64];
    }
    __syncthreads();
    if ((tid & 0x7F) < 32)
    {
        temp_sum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (uint8_t offset = 32 / 2; offset > 0; offset = offset >> 1)
        {
            temp_sum += __shfl_down_sync(FULL_MASK, temp_sum, offset);
        }
    }
    __syncthreads();

    if (tid == 0)
    {
        // image_sum
        contrast_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 128)
    {
        // image_sum_del_theta_x
        contrast_del_x_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 256)
    {
        // image_sum_del_theta_y
        contrast_del_y_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 384)
    {
        // image_sum_del_theta_x
        contrast_del_z_block_sum[blockIdx.x] = temp_sum;
    }
}
void fillImageBilinear(float fx, float fy, float cx, float cy, int height, int width, int num_events, float *x_unprojected, float *y_unprojected, float *x_prime, float *y_prime, float *t, float *image, const float rotation_x, const float rotation_y, const float rotation_z, float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum)
{
    // const int num_sm = 8; // Jetson Orin NX
    // const int blocks_per_sm = 4;
    // const int threads_per_block = 128;
    int blockSize = 512; // The launch configurator returned block size
    // int minGridSize; // The minimum grid size needed to achieve the
    // maximum occupancy for a full device launch
    int gridSize; // The actual grid size needed, based on input size

    // hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
    //                                    fillImageBilinear_, 0, 0);
    // Round up according to array size
    gridSize = (num_events + blockSize - 1) / blockSize;

    int smemSize = blockSize * sizeof(float);
    fillImageBilinear_<<<gridSize, blockSize, smemSize>>>(fx, fy, cx, cy, height, width, num_events, x_unprojected, y_unprojected, x_prime, y_prime, t, image, rotation_x, rotation_y, rotation_z, contrast_block_sum, contrast_del_x_block_sum, contrast_del_y_block_sum, contrast_del_z_block_sum);
}

__global__ void fillImageKronecker_(int height, int width, int num_events, float *x_prime, float *y_prime, float *image)
{

    size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);

    for (size_t i = thread_grid_idx; i < num_events; i += num_threads_in_grid)
    {
        // populate image
        // check if coordinates are 3 pixels in of the boundary
        int x_round = round(x_prime[i]);
        int y_round = round(y_prime[i]);
        if (x_round >= 1 && x_round <= width && y_round >= 1 && y_round <= height)
        {
            int idx = (y_round - 1) * width + x_round - 1;
            atomicAdd(&image[idx], 1);
        }
    }
}
void fillImageKronecker(int height, int width, int num_events, float *x_prime, float *y_prime, float *image)
{
    const int num_sm = 8; // Jetson Orin NX
    const int blocks_per_sm = 4;
    const int threads_per_block = 128;
    hipMemset(image, 0, height * width * sizeof(float));
    fillImageKronecker_<<<blocks_per_sm * num_sm, threads_per_block>>>(height, width, num_events, x_prime, y_prime, image);
}
float getMax(float *image, int height, int width)
{
    float *out;
    hipMalloc(&out, sizeof(float));
    size_t temp_cub_temp_size;
    float *temp_storage = NULL;
    hipcub::DeviceReduce::Reduce(temp_storage, temp_cub_temp_size, image, out, (height) * (width), hipcub::Max(), 0);
    hipDeviceSynchronize();
    hipMalloc(&temp_storage, temp_cub_temp_size);
    hipcub::DeviceReduce::Reduce(temp_storage, temp_cub_temp_size, image, out, (height) * (width), hipcub::Max(), 0);
    hipDeviceSynchronize();
    float maximum;
    hipMemcpy(&maximum, out, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(out);
    hipFree(temp_storage);
    return maximum;
}

__global__ void getContrastDelBatchReduceHarder_(float *image, int num_elements, float *means, float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum, int prev_gridsize)
{

    float *image_del_x = image + num_elements;
    float *image_del_y = image + num_elements * 2;
    float *image_del_z = image + num_elements * 3;
    // START COPY
    float *sdata = SharedMemory<float>();
    float temp_sum = 0;
    uint16_t tid = threadIdx.x;
    // 85 partial sums to go
    // dump partial sums inside again
    if (tid < prev_gridsize)
    {

        if (blockIdx.x == 0)
        {
            temp_sum = contrast_block_sum[tid];
        }
        else if (blockIdx.x == 1)
        {
            temp_sum = contrast_del_x_block_sum[tid];
        }
        else if (blockIdx.x == 2)
        {
            temp_sum = contrast_del_y_block_sum[tid];
        }
        else if (blockIdx.x == 3)
        {
            temp_sum = contrast_del_z_block_sum[tid];
        }
    }
    sdata[tid] = temp_sum;
    __syncthreads();
    if ((tid) < 64)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 64];
    }
    __syncthreads();
    if ((tid) < 32)
    {
        temp_sum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (uint8_t offset = 32 / 2; offset > 0; offset = offset >> 1)
        {
            temp_sum += __shfl_down_sync(FULL_MASK, temp_sum, offset);
        }
    }
    if (tid == 0)
    {
        if (blockIdx.x == 0)
        {
            means[0] = temp_sum / num_elements;
        }
        else if (blockIdx.x == 1)
        {
            means[1] = temp_sum / num_elements;
        }
        else if (blockIdx.x == 2)
        {
            means[2] = temp_sum / num_elements;
        }
        else
        {
            means[3] = temp_sum / num_elements;
        }
    }

    // END COPY
    float image_contrast = 0;
    float image_contrast_del_theta_x = 0;
    float image_contrast_del_theta_y = 0;
    float image_contrast_del_theta_z = 0;
    size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    // size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);
    size_t idx = thread_grid_idx;
    __syncthreads();
    while (idx < num_elements)
    {
        float image_norm = image[idx] - means[0];
        float image_norm_x = image_del_x[idx] - means[1];
        float image_norm_y = image_del_y[idx] - means[2];
        float image_norm_z = image_del_z[idx] - means[3];
        image_contrast = image_norm * image_norm;
        image_contrast_del_theta_x = image_norm * image_norm_x;
        image_contrast_del_theta_y = image_norm * image_norm_y;
        image_contrast_del_theta_z = image_norm * image_norm_z;
        idx += blockDim.x * gridDim.x;
    }
    // float *sdata = SharedMemory<float>();
    // uint16_t tid = threadIdx.x;

    // do reduction in shared mem

    // sum up to 128 elements

    // float temp_sum;
    // image_contrast
    sdata[tid] = image_contrast;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_contrast = image_contrast + sdata[tid + 256];
    __syncthreads();
    // store contrast in 0 to 127
    if (tid < 128)
        temp_sum = image_contrast + sdata[tid + 128];
    __syncthreads();
    // image_contrast_del_theta_x
    sdata[tid] = image_contrast_del_theta_x;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_contrast_del_theta_x = image_contrast_del_theta_x + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
        sdata[tid] = image_contrast_del_theta_x = image_contrast_del_theta_x + sdata[tid + 128];
    __syncthreads();
    // store x in 128 to 255
    if (tid >= 128 && tid < 256)
    {
        temp_sum = sdata[tid - 128];
    }
    __syncthreads();
    // image_contrast_del_theta_y
    sdata[tid] = image_contrast_del_theta_y;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_contrast_del_theta_y = image_contrast_del_theta_y + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
        image_contrast_del_theta_y = image_contrast_del_theta_y + sdata[tid + 128];
    __syncthreads();
    // store y in 256 to 383
    if (tid >= 256 && tid < 384)
    {
        temp_sum = sdata[tid - 256];
    }
    __syncthreads();
    // image_contrast_del_theta_z
    sdata[tid] = image_contrast_del_theta_z;
    __syncthreads();
    if (tid < 256)
        sdata[tid] = image_contrast_del_theta_z = image_contrast_del_theta_z + sdata[tid + 256];
    __syncthreads();
    if (tid < 128)
    {
        sdata[tid] = image_contrast_del_theta_z = image_contrast_del_theta_z + sdata[tid + 128];
    }
    __syncthreads();
    // store z in 384 to 512
    if (tid >= 384)
    {
        temp_sum = sdata[tid - 384];
    }
    // dump partial sums inside again
    sdata[tid] = temp_sum;
    __syncthreads();
    if ((tid & 0x7F) < 64)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 64];
    }
    __syncthreads();
    if ((tid & 0x7F) < 32)
    {
        temp_sum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (uint8_t offset = 32 / 2; offset > 0; offset = offset >> 1)
        {
            temp_sum += __shfl_down_sync(FULL_MASK, temp_sum, offset);
        }
    }
    __syncthreads();

    if (tid == 0)
    {
        // image_contrast
        contrast_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 128)
    {
        // image_contrast_del_theta_x
        contrast_del_x_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 256)
    {
        // image_contrast_del_theta_y
        contrast_del_y_block_sum[blockIdx.x] = temp_sum;
    }
    else if (tid == 384)
    {
        // image_contrast_del_theta_x
        contrast_del_z_block_sum[blockIdx.x] = temp_sum;
    }
}

// 4 blocks x threads
template <int prev_gridsize>
__global__ void getContrastDelBatchReduceHarderPt2_(float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum)
{
    float *sdata = SharedMemory<float>();
    float temp_sum;
    uint16_t tid = threadIdx.x;
    // 85 partial sums to go
    // dump partial sums inside again
    if (tid < prev_gridsize)
    {

        if (blockIdx.x == 0)
        {
            temp_sum = temp_sum = contrast_block_sum[tid];
        }
        else if (blockIdx.x == 1)
        {
            temp_sum = contrast_del_x_block_sum[tid];
        }
        else if (blockIdx.x == 2)
        {
            temp_sum = contrast_del_y_block_sum[tid];
        }
        else
        {
            temp_sum = contrast_del_z_block_sum[tid];
        }
    }
    else
    {
        temp_sum = 0;
    }
    sdata[tid] = temp_sum;
    __syncthreads();

    if (prev_gridsize > 256 && (tid) < 256)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 256];
    }
    __syncthreads();
    if (prev_gridsize > 128 && (tid) < 128)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 128];
    }
    __syncthreads();
    if (prev_gridsize > 64 && (tid) < 64)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 64];
    }
    __syncthreads();
    if ((tid) < 32)
    {
        temp_sum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (uint8_t offset = 32 / 2; offset > 0; offset = offset >> 1)
        {
            temp_sum += __shfl_down_sync(FULL_MASK, temp_sum, offset);
        }
    }
    if (tid == 0)
    {
        if (blockIdx.x == 0)
        {
            contrast_block_sum[0] = temp_sum;
        }
        else if (blockIdx.x == 1)
        {
            contrast_block_sum[1] = temp_sum;
        }
        else if (blockIdx.x == 2)
        {
            contrast_block_sum[2] = temp_sum;
        }
        else
        {
            contrast_block_sum[3] = temp_sum;
        }
    }
}

// 4 blocks 128 threads
__global__ void meanPt2_(float *contrast_block_sum, float *contrast_del_x_block_sum, float *contrast_del_y_block_sum, float *contrast_del_z_block_sum, int num_elements, float *means, int prev_gridsize)
{
    float *sdata = SharedMemory<float>();
    float temp_sum = 0;
    uint16_t tid = threadIdx.x;
    // 85 partial sums to go
    // dump partial sums inside again
    if (tid < prev_gridsize)
    {

        if (blockIdx.x == 0)
        {
            temp_sum = contrast_block_sum[tid];
        }
        else if (blockIdx.x == 1)
        {
            temp_sum = contrast_del_x_block_sum[tid];
        }
        else if (blockIdx.x == 2)
        {
            temp_sum = contrast_del_y_block_sum[tid];
        }
        else
        {
            temp_sum = contrast_del_z_block_sum[tid];
        }
    }
    sdata[tid] = temp_sum;
    __syncthreads();
    if ((tid) < 64)
    {
        sdata[tid] = temp_sum = temp_sum + sdata[tid + 64];
    }
    __syncthreads();
    if ((tid) < 32)
    {
        temp_sum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (uint8_t offset = 32 / 2; offset > 0; offset = offset >> 1)
        {
            temp_sum += __shfl_down_sync(FULL_MASK, temp_sum, offset);
        }
    }
    if (tid == 0)
    {
        if (blockIdx.x == 0)
        {
            means[0] = temp_sum / num_elements;
        }
        else if (blockIdx.x == 1)
        {
            means[1] = temp_sum / num_elements;
        }
        else if (blockIdx.x == 2)
        {
            means[2] = temp_sum / num_elements;
        }
        else
        {
            means[3] = temp_sum / num_elements;
        }
    }
}

void getContrastDelBatchReduce(float *image,
                               double *image_contrast, double *image_del_theta_contrast,
                               int height, int width,
                               float *contrast_block_sum,
                               float *contrast_del_x_block_sum,
                               float *contrast_del_y_block_sum,
                               float *contrast_del_z_block_sum,
                               float *means,
                               int num_events,
                               hipStream_t const *stream)
{
    int blockSize = 512; // The launch configurator returned block size
    int prev_gridsize = (num_events + blockSize - 1) / blockSize;
    // int gridSize = 85; // The actual grid size needed, based on input size
    int gridSize = std::min(512, (height * width + blockSize - 1) / blockSize);

    int smemSize = (blockSize <= 32) ? 2 * blockSize * sizeof(float) : blockSize * sizeof(float);

    // meanPt2_<<<4, 128, 128 * sizeof(float)>>>(contrast_block_sum, contrast_del_x_block_sum, contrast_del_y_block_sum, contrast_del_z_block_sum, height * width, means,prev_gridsize);

    getContrastDelBatchReduceHarder_<<<gridSize, blockSize, smemSize>>>(image, height * width, means, contrast_block_sum, contrast_del_x_block_sum, contrast_del_y_block_sum, contrast_del_z_block_sum, prev_gridsize);
    if (height == 180 && width == 240)
        getContrastDelBatchReduceHarderPt2_<85><<<4, 512, 512 * sizeof(float), stream[0]>>>(contrast_block_sum, contrast_del_x_block_sum, contrast_del_y_block_sum, contrast_del_z_block_sum);
    else if (height == 480 && width == 640)
        getContrastDelBatchReduceHarderPt2_<512><<<4, 512, 512 * sizeof(float), stream[0]>>>(contrast_block_sum, contrast_del_x_block_sum, contrast_del_y_block_sum, contrast_del_z_block_sum);

    hipMemsetAsync(image, 0, (height) * (width) * sizeof(float) * 4, stream[1]);
    hipDeviceSynchronize();
    {

        nvtx3::scoped_range r{"final contrast"};
        int num_el = height * width;
        image_contrast[0] = -contrast_block_sum[0] / num_el;
        image_del_theta_contrast[0] = -2 * contrast_block_sum[1] / num_el;
        image_del_theta_contrast[1] = -2 * contrast_block_sum[2] / num_el;
        image_del_theta_contrast[2] = -2 * contrast_block_sum[3] / num_el;
    }
}

__device__ float getRandom(uint64_t seed, int tid, int threadCallCount)
{
    hiprandState s;
    hiprand_init(seed + tid + threadCallCount, 0, 0, &s);
    // return hiprand_uniform(&s);
    return hiprand_log_normal(&s, 1e-16, 10.0);
}
__global__ void one_step_kernel_(uint64_t seed, float *randoms, int numel)
{
    size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);
    for (size_t idx = thread_grid_idx; idx < numel; idx += num_threads_in_grid)
    {

        randoms[idx] = getRandom(seed, idx, 0);
    }
}

void one_step_kernel(uint64_t seed, float *randoms, int numel)
{
    one_step_kernel_<<<43, 1024>>>(seed, randoms, numel);
}