#include "hip/hip_runtime.h"
#include "motion_compensation.h"

#include <cmath>
#include <algorithm> //for std::max
#include <cstdio>
#include <vector>
#include <iostream>
#include "utils.h"

#include <hipcub/hipcub.hpp>
#include <cub/block/block_reduce.cuh>

#include <jetson-utils/cudaMappedMemory.h>

__global__ void fillImage_(float fx, float fy, float cx, float cy, int height, int width, int num_events, float *x_unprojected, float *y_unprojected, float *x_prime, float *y_prime, float *t, float *image, const float rotation_x, const float rotation_y, const float rotation_z, bool do_jacobian, float *image_del_x, float *image_del_y, float *image_del_z)
{

    size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);

    for (size_t i = thread_grid_idx; i < num_events; i += num_threads_in_grid)
    {
        float gaussian;
        // calculate theta x,y,z
        float theta_x_t = rotation_x * t[i];
        float theta_y_t = rotation_y * t[i];
        float theta_z_t = rotation_z * t[i];

        // calculate x/y/z_rotated
        float z_rotated_inv = 1 / (-theta_y_t * x_unprojected[i] + theta_x_t * y_unprojected[i] + 1);
        float x_rotated_norm = (x_unprojected[i] - theta_z_t * y_unprojected[i] + theta_y_t) * z_rotated_inv;
        float y_rotated_norm = (theta_z_t * x_unprojected[i] + y_unprojected[i] - theta_x_t) * z_rotated_inv;

        // calculate x_prime and y_prime
        x_prime[i] = fx * x_rotated_norm + cx;
        y_prime[i] = fy * y_rotated_norm + cy;

        float del_x_del_theta_x, del_x_del_theta_y, del_x_del_theta_z, del_y_del_theta_x, del_y_del_theta_y, del_y_del_theta_z;
        if (do_jacobian)
        {
            float fx_div_z_rotated = fx * z_rotated_inv;
            float fy_div_z_rotated = fy * z_rotated_inv;
            del_x_del_theta_y = fx_div_z_rotated * t[i] * (1 + x_unprojected[i] * x_rotated_norm);
            del_x_del_theta_z = fx_div_z_rotated * (-t[i] * y_unprojected[i]);
            del_x_del_theta_x = del_x_del_theta_z * x_rotated_norm;
            del_y_del_theta_x = fy_div_z_rotated * t[i] * (-1 - y_unprojected[i] * y_rotated_norm);
            del_y_del_theta_z = fy_div_z_rotated * (t[i] * x_unprojected[i]);
            del_y_del_theta_y = del_y_del_theta_z * y_rotated_norm;
        }
        // populate image

        // Pseudo Gaussian
        // check if coordinates are 3 pixels in of the boundary
        int x_round = round(x_prime[i]);
        int y_round = round(y_prime[i]);
        if (x_round >= 1 && x_round <= width && y_round >= 1 && y_round <= height)
        {
            for (int row = y_round - 3; row < y_round + 4; row++)
            {
                for (int col = x_round - 3; col < x_round + 4; col++)
                {
                    // TODO: make a LUT for the values here rounded to a certain s.f. and see if there is a speed-up
                    float x_diff = col - x_prime[i];
                    float y_diff = row - y_prime[i];
                    // float x_diff = col - x_unprojected[i];
                    // float y_diff = row - y_unprojected[i];
                    gaussian = exp((-x_diff * x_diff - y_diff * y_diff) / 2) / sqrt(2 * M_PI);
                    int idx = (row + 3 - 1) * (width + 6) + col + 3 - 1;
                    atomicAdd(&image[idx], gaussian);
                    if (do_jacobian)
                    {
                        atomicAdd(&image_del_x[idx], gaussian * (x_diff * del_x_del_theta_x + y_diff * del_y_del_theta_x));
                        atomicAdd(&image_del_y[idx], gaussian * (x_diff * del_x_del_theta_y + y_diff * del_y_del_theta_y));
                        atomicAdd(&image_del_z[idx], gaussian * (x_diff * del_x_del_theta_z + y_diff * del_y_del_theta_z));
                    }
                }
            }
        }
    }
}

void fillImage(float fx, float fy, float cx, float cy, int height, int width, int num_events, float *x_unprojected, float *y_unprojected, float *x_prime, float *y_prime, float *t, float *image, const float rotation_x, const float rotation_y, const float rotation_z, bool do_jacobian, float *image_del_x, float *image_del_y, float *image_del_z)
{
    const int num_sm = 8; // Jetson Orin NX
    const int blocks_per_sm = 4;
    const int threads_per_block = 128;
    fillImage_<<<blocks_per_sm * num_sm, threads_per_block>>>(fx, fy, cx, cy, height, width, num_events, x_unprojected, y_unprojected, x_prime, y_prime, t, image, rotation_x, rotation_y, rotation_z, do_jacobian, image_del_x, image_del_y, image_del_z);
}
__global__ void fillImageBilinearIntrinsics_(float fx, float fy, float cx, float cy, int height, int width, int num_events, float *x_unprojected, float *y_unprojected, float *x_prime, float *y_prime, float *t, float *image, const float rotation_x, const float rotation_y, const float rotation_z, bool do_jacobian, float *image_del_x, float *image_del_y, float *image_del_z)
{

    size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);

    for (size_t i = thread_grid_idx; i < num_events; i += num_threads_in_grid)
    {
        // calculate theta x,y,z
        float theta_x_t = __fmul_rn(rotation_x, t[i]);
        float theta_y_t = __fmul_rn(rotation_y, t[i]);
        float theta_z_t = __fmul_rn(rotation_z, t[i]);

        // calculate x/y/z_rotated
        float z_rotated_inv = __fdiv_rn(1, (__fmul_rn(-theta_y_t, x_unprojected[i]) + __fmaf_rn(theta_x_t, y_unprojected[i], 1)));
        float x_rotated_norm = __fmul_rn(__fmaf_rn(-theta_z_t, y_unprojected[i], __fadd_rn(x_unprojected[i], theta_y_t)), z_rotated_inv);
        float y_rotated_norm = __fmul_rn(__fmaf_rn(theta_z_t, x_unprojected[i], __fsub_rn(y_unprojected[i], theta_x_t)), z_rotated_inv);

        // calculate x_prime and y_prime
        x_prime[i] = __fmaf_rn(fx, x_rotated_norm, cx);
        y_prime[i] = __fmaf_rn(fy, y_rotated_norm, cy);

        float del_x_del_theta_x, del_x_del_theta_y, del_x_del_theta_z, del_y_del_theta_x, del_y_del_theta_y, del_y_del_theta_z;
        // populate image

        // Bilinear
        // check if coordinates are 3 pixels in of the boundary
        int x_trunc = int(x_prime[i]);
        int y_trunc = int(y_prime[i]);
        if (x_trunc >= 1 && x_trunc <= width - 2 && y_trunc >= 1 && y_trunc <= height - 2)
        {
            int idx1 = __fmaf_rn(__fsub_rn(y_trunc, 1), width, __fsub_rn(x_trunc, 1));
            int idx2 = __fmaf_rn(__fsub_rn(y_trunc, 1), width, x_trunc);
            int idx3 = __fmaf_rn((y_trunc), width, __fsub_rn(x_trunc, 1));
            int idx4 = __fmaf_rn((y_trunc), width, x_trunc);
            float x_diff = __fsub_rn(x_prime[i], x_trunc);
            float y_diff = __fsub_rn(y_prime[i], y_trunc);
            atomicAdd(&image[idx1], __fmul_rn(__fsub_rn(1, x_diff), __fsub_rn(1, y_diff)));
            atomicAdd(&image[idx2], __fmul_rn((x_diff), __fsub_rn(1, y_diff)));
            atomicAdd(&image[idx3], __fsub_rn(1, x_diff) * (y_diff));
            atomicAdd(&image[idx4], __fmul_rn((x_diff), (y_diff)));

            if (do_jacobian)
            {
                float fx_div_z_rotated = __fmul_rn(fx, z_rotated_inv);
                float fy_div_z_rotated = __fmul_rn(fy, z_rotated_inv);
                del_x_del_theta_y = __fmul_rn(__fmul_rn(fx_div_z_rotated, t[i]), __fmaf_rn(x_unprojected[i], x_rotated_norm, 1));
                del_x_del_theta_z = __fmul_rn(fx_div_z_rotated, __fmul_rn(-t[i], y_unprojected[i]));
                del_x_del_theta_x = __fmul_rn(del_x_del_theta_z, x_rotated_norm);
                del_y_del_theta_x = __fmul_rn(__fmul_rn(fy_div_z_rotated, t[i]), __fmaf_rn(-y_unprojected[i], y_rotated_norm, -1));
                del_y_del_theta_z = __fmul_rn(fy_div_z_rotated, __fmul_rn(t[i], x_unprojected[i]));
                del_y_del_theta_y = __fmul_rn(del_y_del_theta_z, y_rotated_norm);
                float d1x = __fsub_rn(y_diff, 1);
                float d1y = __fsub_rn(x_diff, 1);
                float d2x = __fsub_rn(1, y_diff);
                float d2y = -x_diff;
                float d3x = -y_diff;
                float d3y = __fsub_rn(1, x_diff);
                float d4x = y_diff;
                float d4y = x_diff;
                atomicAdd(&image_del_x[idx1], __fmaf_rn(d1x, del_x_del_theta_x, __fmul_rn(d1y, del_y_del_theta_x)));
                atomicAdd(&image_del_x[idx2], __fmaf_rn(d2x, del_x_del_theta_x, __fmul_rn(d2y, del_y_del_theta_x)));
                atomicAdd(&image_del_x[idx3], __fmaf_rn(d3x, del_x_del_theta_x, __fmul_rn(d3y, del_y_del_theta_x)));
                atomicAdd(&image_del_x[idx4], __fmaf_rn(d4x, del_x_del_theta_x, __fmul_rn(d4y, del_y_del_theta_x)));

                atomicAdd(&image_del_y[idx1], __fmaf_rn(d1x, del_x_del_theta_y, __fmul_rn(d1y, del_y_del_theta_y)));
                atomicAdd(&image_del_y[idx2], __fmaf_rn(d2x, del_x_del_theta_y, __fmul_rn(d2y, del_y_del_theta_y)));
                atomicAdd(&image_del_y[idx3], __fmaf_rn(d3x, del_x_del_theta_y, __fmul_rn(d3y, del_y_del_theta_y)));
                atomicAdd(&image_del_y[idx4], __fmaf_rn(d4x, del_x_del_theta_y, __fmul_rn(d4y, del_y_del_theta_y)));

                atomicAdd(&image_del_z[idx1], __fmaf_rn(d1x, del_x_del_theta_z, __fmul_rn(d1y, del_y_del_theta_z)));
                atomicAdd(&image_del_z[idx2], __fmaf_rn(d2x, del_x_del_theta_z, __fmul_rn(d2y, del_y_del_theta_z)));
                atomicAdd(&image_del_z[idx3], __fmaf_rn(d3x, del_x_del_theta_z, __fmul_rn(d3y, del_y_del_theta_z)));
                atomicAdd(&image_del_z[idx4], __fmaf_rn(d4x, del_x_del_theta_z, __fmul_rn(d4y, del_y_del_theta_z)));
            }
        }
    }
}
void fillImageBilinearIntrinsics(float fx, float fy, float cx, float cy, int height, int width, int num_events, float *x_unprojected, float *y_unprojected, float *x_prime, float *y_prime, float *t, float *image, const float rotation_x, const float rotation_y, const float rotation_z, bool do_jacobian, float *image_del_x, float *image_del_y, float *image_del_z)
{
    // const int num_sm = 8; // Jetson Orin NX
    // const int blocks_per_sm = 4;
    // const int threads_per_block = 128;
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
                     // maximum occupancy for a full device launch
    int gridSize;    // The actual grid size needed, based on input size

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                       fillImageBilinearIntrinsics_, 0, 0);
    // Round up according to array size
    gridSize = (num_events + blockSize - 1) / blockSize;
    fillImageBilinearIntrinsics_<<<gridSize, blockSize>>>(fx, fy, cx, cy, height, width, num_events, x_unprojected, y_unprojected, x_prime, y_prime, t, image, rotation_x, rotation_y, rotation_z, do_jacobian, image_del_x, image_del_y, image_del_z);
}

__global__ void fillImageBilinear_(float fx, float fy, float cx, float cy, int height, int width, int num_events, float *x_unprojected, float *y_unprojected, float *x_prime, float *y_prime, float *t, float *image, const float rotation_x, const float rotation_y, const float rotation_z, bool do_jacobian, float *image_del_x, float *image_del_y, float *image_del_z)
{

    size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);

    for (size_t i = thread_grid_idx; i < num_events; i += num_threads_in_grid)
    {
        // calculate theta x,y,z
        float theta_x_t = rotation_x * t[i];
        float theta_y_t = rotation_y * t[i];
        float theta_z_t = rotation_z * t[i];

        // calculate x/y/z_rotated
        float z_rotated_inv = 1 / (-theta_y_t * x_unprojected[i] + theta_x_t * y_unprojected[i] + 1);
        float x_rotated_norm = (x_unprojected[i] - theta_z_t * y_unprojected[i] + theta_y_t) * z_rotated_inv;
        float y_rotated_norm = (theta_z_t * x_unprojected[i] + y_unprojected[i] - theta_x_t) * z_rotated_inv;

        // calculate x_prime and y_prime
        x_prime[i] = fx * x_rotated_norm + cx;
        y_prime[i] = fy * y_rotated_norm + cy;

        // populate image

        // Bilinear
        // check if coordinates are 3 pixels in of the boundary
        int x_trunc = int(x_prime[i]);
        int y_trunc = int(y_prime[i]);
        if (x_trunc >= 1 && x_trunc <= width - 2 && y_trunc >= 1 && y_trunc <= height - 2)
        {

            int idx1 = x_trunc - 1 + (y_trunc - 1) * width;
            int idx2 = x_trunc + (y_trunc - 1) * width;
            int idx3 = x_trunc - 1 + (y_trunc)*width;
            int idx4 = x_trunc + (y_trunc)*width;
            float x_diff = x_prime[i] - x_trunc;
            float y_diff = y_prime[i] - y_trunc;
            atomicAdd(&image[idx1], (1 - x_diff) * (1 - y_diff));
            atomicAdd(&image[idx2], (x_diff) * (1 - y_diff));
            atomicAdd(&image[idx3], (1 - x_diff) * (y_diff));
            atomicAdd(&image[idx4], (x_diff) * (y_diff));
            if (do_jacobian)
            {

                float del_x_del_theta_x, del_x_del_theta_y, del_x_del_theta_z, del_y_del_theta_x, del_y_del_theta_y, del_y_del_theta_z;
                float fx_div_z_rotated = fx * z_rotated_inv;
                float fy_div_z_rotated = fy * z_rotated_inv;
                del_x_del_theta_y = fx_div_z_rotated * t[i] * (1 + x_unprojected[i] * x_rotated_norm);
                del_x_del_theta_z = fx_div_z_rotated * (-t[i] * y_unprojected[i]);
                del_x_del_theta_x = del_x_del_theta_z * x_rotated_norm;
                del_y_del_theta_x = fy_div_z_rotated * t[i] * (-1 - y_unprojected[i] * y_rotated_norm);
                del_y_del_theta_z = fy_div_z_rotated * (t[i] * x_unprojected[i]);
                del_y_del_theta_y = del_y_del_theta_z * y_rotated_norm;
                float d1x = -(1 - y_diff);
                float d1y = -(1 - x_diff);
                float d2x = 1 - y_diff;
                float d2y = -x_diff;
                float d3x = -y_diff;
                float d3y = 1 - x_diff;
                float d4x = y_diff;
                float d4y = x_diff;
                atomicAdd(&image_del_x[idx1], d1x * del_x_del_theta_x + d1y * del_y_del_theta_x);
                atomicAdd(&image_del_x[idx2], d2x * del_x_del_theta_x + d2y * del_y_del_theta_x);
                atomicAdd(&image_del_x[idx3], d3x * del_x_del_theta_x + d3y * del_y_del_theta_x);
                atomicAdd(&image_del_x[idx4], d4x * del_x_del_theta_x + d4y * del_y_del_theta_x);

                atomicAdd(&image_del_y[idx1], d1x * del_x_del_theta_y + d1y * del_y_del_theta_y);
                atomicAdd(&image_del_y[idx2], d2x * del_x_del_theta_y + d2y * del_y_del_theta_y);
                atomicAdd(&image_del_y[idx3], d3x * del_x_del_theta_y + d3y * del_y_del_theta_y);
                atomicAdd(&image_del_y[idx4], d4x * del_x_del_theta_y + d4y * del_y_del_theta_y);

                atomicAdd(&image_del_z[idx1], d1x * del_x_del_theta_z + d1y * del_y_del_theta_z);
                atomicAdd(&image_del_z[idx2], d2x * del_x_del_theta_z + d2y * del_y_del_theta_z);
                atomicAdd(&image_del_z[idx3], d3x * del_x_del_theta_z + d3y * del_y_del_theta_z);
                atomicAdd(&image_del_z[idx4], d4x * del_x_del_theta_z + d4y * del_y_del_theta_z);
            }
        }
    }
}
void fillImageBilinear(float fx, float fy, float cx, float cy, int height, int width, int num_events, float *x_unprojected, float *y_unprojected, float *x_prime, float *y_prime, float *t, float *image, const float rotation_x, const float rotation_y, const float rotation_z, bool do_jacobian, float *image_del_x, float *image_del_y, float *image_del_z)
{
    // const int num_sm = 8; // Jetson Orin NX
    // const int blocks_per_sm = 4;
    // const int threads_per_block = 128;
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
                     // maximum occupancy for a full device launch
    int gridSize;    // The actual grid size needed, based on input size

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                       fillImageBilinear_, 0, 0);
    // Round up according to array size
    gridSize = (num_events + blockSize - 1) / blockSize;
    fillImageBilinear_<<<gridSize, blockSize>>>(fx, fy, cx, cy, height, width, num_events, x_unprojected, y_unprojected, x_prime, y_prime, t, image, rotation_x, rotation_y, rotation_z, do_jacobian, image_del_x, image_del_y, image_del_z);
}

__global__ void warpBilinear_(float fx, float fy, float cx, float cy, int height, int width, int num_events, float *x_unprojected, float *y_unprojected, float *x_prime, float *y_prime, float *t, float *image, const float rotation_x, const float rotation_y, const float rotation_z, bool do_jacobian, float *image_del_x, float *image_del_y, float *image_del_z, float *z_rotated_inv, float *x_rotated_norm, float *y_rotated_norm)
{

    size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);

    for (size_t i = thread_grid_idx; i < num_events; i += num_threads_in_grid)
    {
        // calculate theta x,y,z
        float theta_x_t = rotation_x * t[i];
        float theta_y_t = rotation_y * t[i];
        float theta_z_t = rotation_z * t[i];

        // calculate x/y/z_rotated
        z_rotated_inv[i] = 1 / (-theta_y_t * x_unprojected[i] + theta_x_t * y_unprojected[i] + 1);
        x_rotated_norm[i] = (x_unprojected[i] - theta_z_t * y_unprojected[i] + theta_y_t) * z_rotated_inv[i];
        y_rotated_norm[i] = (theta_z_t * x_unprojected[i] + y_unprojected[i] - theta_x_t) * z_rotated_inv[i];

        // calculate x_prime and y_prime
        x_prime[i] = fx * x_rotated_norm[i] + cx;
        y_prime[i] = fy * y_rotated_norm[i] + cy;
    }
}

__global__ void fillImageOnly_(float fx, float fy, float cx, float cy, int height, int width, int num_events, float *x_unprojected, float *y_unprojected, float *x_prime, float *y_prime, float *t, float *image, const float rotation_x, const float rotation_y, const float rotation_z, bool do_jacobian, float *image_del_x, float *image_del_y, float *image_del_z, float *z_rotated_inv, float *x_rotated_norm, float *y_rotated_norm)
{

    size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);

    for (size_t i = thread_grid_idx; i < num_events; i += num_threads_in_grid)
    {
        // Bilinear
        // check if coordinates are 3 pixels in of the boundary
        int x_trunc = int(x_prime[i]);
        int y_trunc = int(y_prime[i]);
        if (x_trunc >= 1 && x_trunc <= width - 2 && y_trunc >= 1 && y_trunc <= height - 2)
        {
            int idx1 = x_trunc - 1 + (y_trunc - 1) * width;
            int idx2 = x_trunc + (y_trunc - 1) * width;
            int idx3 = x_trunc - 1 + (y_trunc)*width;
            int idx4 = x_trunc + (y_trunc)*width;
            float x_diff = x_prime[i] - x_trunc;
            float y_diff = y_prime[i] - y_trunc;
            atomicAdd(&image[idx1], (1 - x_diff) * (1 - y_diff));
            atomicAdd(&image[idx2], (x_diff) * (1 - y_diff));
            atomicAdd(&image[idx3], (1 - x_diff) * (y_diff));
            atomicAdd(&image[idx4], (x_diff) * (y_diff));

            if (do_jacobian)
            {

                float del_x_del_theta_x, del_x_del_theta_y, del_x_del_theta_z, del_y_del_theta_x, del_y_del_theta_y, del_y_del_theta_z;
                float fx_div_z_rotated = fx * z_rotated_inv[i];
                float fy_div_z_rotated = fy * z_rotated_inv[i];
                del_x_del_theta_y = fx_div_z_rotated * t[i] * (1 + x_unprojected[i] * x_rotated_norm[i]);
                del_x_del_theta_z = fx_div_z_rotated * (-t[i] * y_unprojected[i]);
                del_x_del_theta_x = del_x_del_theta_z * x_rotated_norm[i];
                del_y_del_theta_x = fy_div_z_rotated * t[i] * (-1 - y_unprojected[i] * y_rotated_norm[i]);
                del_y_del_theta_z = fy_div_z_rotated * (t[i] * x_unprojected[i]);
                del_y_del_theta_y = del_y_del_theta_z * y_rotated_norm[i];
                float d1x = -(1 - y_diff);
                float d1y = -(1 - x_diff);
                float d2x = 1 - y_diff;
                float d2y = -x_diff;
                float d3x = -y_diff;
                float d3y = 1 - x_diff;
                float d4x = y_diff;
                float d4y = x_diff;
                atomicAdd(&image_del_x[idx1], d1x * del_x_del_theta_x + d1y * del_y_del_theta_x);
                atomicAdd(&image_del_x[idx2], d2x * del_x_del_theta_x + d2y * del_y_del_theta_x);
                atomicAdd(&image_del_x[idx3], d3x * del_x_del_theta_x + d3y * del_y_del_theta_x);
                atomicAdd(&image_del_x[idx4], d4x * del_x_del_theta_x + d4y * del_y_del_theta_x);

                atomicAdd(&image_del_y[idx1], d1x * del_x_del_theta_y + d1y * del_y_del_theta_y);
                atomicAdd(&image_del_y[idx2], d2x * del_x_del_theta_y + d2y * del_y_del_theta_y);
                atomicAdd(&image_del_y[idx3], d3x * del_x_del_theta_y + d3y * del_y_del_theta_y);
                atomicAdd(&image_del_y[idx4], d4x * del_x_del_theta_y + d4y * del_y_del_theta_y);

                atomicAdd(&image_del_z[idx1], d1x * del_x_del_theta_z + d1y * del_y_del_theta_z);
                atomicAdd(&image_del_z[idx2], d2x * del_x_del_theta_z + d2y * del_y_del_theta_z);
                atomicAdd(&image_del_z[idx3], d3x * del_x_del_theta_z + d3y * del_y_del_theta_z);
                atomicAdd(&image_del_z[idx4], d4x * del_x_del_theta_z + d4y * del_y_del_theta_z);
            }
        }
    }
}
void fillImageBilinearSeparate(float fx, float fy, float cx, float cy, int height, int width, int num_events, float *x_unprojected, float *y_unprojected, float *x_prime, float *y_prime, float *t, float *image, const float rotation_x, const float rotation_y, const float rotation_z, bool do_jacobian, float *image_del_x, float *image_del_y, float *image_del_z)
{

    // const int num_sm = 8; // Jetson Orin NX
    // const int blocks_per_sm = 4;
    // const int threads_per_block = 128;
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
                     // maximum occupancy for a full device launch
    int gridSize;    // The actual grid size needed, based on input size

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                       warpBilinear_, 0, 0);
    // Round up according to array size
    gridSize = (num_events + blockSize - 1) / blockSize;
    float *z_rotated_inv;
    float *x_rotated_norm;
    float *y_rotated_norm;

    hipMalloc(&z_rotated_inv, sizeof(float) * num_events);
    hipMalloc(&x_rotated_norm, sizeof(float) * num_events);
    hipMalloc(&y_rotated_norm, sizeof(float) * num_events);
    warpBilinear_<<<gridSize, blockSize>>>(fx, fy, cx, cy, height, width, num_events, x_unprojected, y_unprojected, x_prime, y_prime, t, image, rotation_x, rotation_y, rotation_z, do_jacobian, image_del_x, image_del_y, image_del_z, z_rotated_inv, x_rotated_norm, y_rotated_norm);

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                       fillImageOnly_, 0, 0);
    // Round up according to array size
    gridSize = (num_events + blockSize - 1) / blockSize;
    fillImageOnly_<<<gridSize, blockSize>>>(fx, fy, cx, cy, height, width, num_events, x_unprojected, y_unprojected, x_prime, y_prime, t, image, rotation_x, rotation_y, rotation_z, do_jacobian, image_del_x, image_del_y, image_del_z, z_rotated_inv, x_rotated_norm, y_rotated_norm);
    checkCudaErrors(hipFree(z_rotated_inv));
    checkCudaErrors(hipFree(x_rotated_norm));
    checkCudaErrors(hipFree(y_rotated_norm));
}
__global__ void fillImageKronecker_(int height, int width, int num_events, float *x_prime, float *y_prime, float *image)
{

    size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);

    for (size_t i = thread_grid_idx; i < num_events; i += num_threads_in_grid)
    {
        // populate image
        // check if coordinates are 3 pixels in of the boundary
        int x_round = round(x_prime[i]);
        int y_round = round(y_prime[i]);
        if (x_round >= 1 && x_round <= width && y_round >= 1 && y_round <= height)
        {
            int idx = (y_round - 1) * width + x_round - 1;
            atomicAdd(&image[idx], 1);
        }
    }
}
void fillImageKronecker(int height, int width, int num_events, float *x_prime, float *y_prime, float *image)
{
    const int num_sm = 8; // Jetson Orin NX
    const int blocks_per_sm = 4;
    const int threads_per_block = 128;
    hipMemset(image, 0, height * width * sizeof(float));
    fillImageKronecker_<<<blocks_per_sm * num_sm, threads_per_block>>>(height, width, num_events, x_prime, y_prime, image);
}
int getCubSize(float *image, int height, int width)
{
    size_t temp_cub_temp_size;
    float *temp_storage = NULL;
    hipcub::DeviceReduce::Reduce(temp_storage, temp_cub_temp_size, image, image, (height) * (width), hipcub::Sum(), 0);
    return temp_cub_temp_size;
}
float getMean(float *image, int height, int width, int cub_temp_size)
{

    float *out;
    hipMalloc(&out, sizeof(float));
    size_t temp_cub_temp_size;
    float *temp_storage = NULL;

    if (cub_temp_size != 0)
    {
        temp_cub_temp_size = cub_temp_size;
    }
    else
    {
        std::cout << "no cub temp size" << std::endl;
        hipcub::DeviceReduce::Reduce(temp_storage, temp_cub_temp_size, image, out, (height) * (width), hipcub::Sum(), 0);
        hipDeviceSynchronize();
    }
    checkCudaErrors(hipMalloc(&temp_storage, temp_cub_temp_size));
    hipcub::DeviceReduce::Reduce(temp_storage, temp_cub_temp_size, image, out, (height) * (width), hipcub::Sum(), 0);
    hipDeviceSynchronize();
    float sum;
    checkCudaErrors(hipMemcpy(&sum, out, sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(out));
    checkCudaErrors(hipFree(temp_storage));

    return sum / ((height) * (width));
}
float getMax(float *image, int height, int width)
{
    float *out;
    hipMalloc(&out, sizeof(float));
    size_t temp_cub_temp_size;
    float *temp_storage = NULL;
    hipcub::DeviceReduce::Reduce(temp_storage, temp_cub_temp_size, image, out, (height) * (width), hipcub::Max(), 0);
    hipDeviceSynchronize();
    hipMalloc(&temp_storage, temp_cub_temp_size);
    hipcub::DeviceReduce::Reduce(temp_storage, temp_cub_temp_size, image, out, (height) * (width), hipcub::Max(), 0);
    hipDeviceSynchronize();
    float maximum;
    hipMemcpy(&maximum, out, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(out);
    hipFree(temp_storage);
    return maximum;
}
float getMeanCpu(float *image, int height, int width)
{

    float sum = 0;
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
            sum += image[i * (width) + j];
    }
    return sum / ((height) * (width));
}
float getMeanCpuCrop(float *image, int height, int width)
{

    float sum = 0;
    for (int i = 3; i < height + 3; i++)
    {
        for (int j = 3; j < width + 3; j++)
        {
            sum += image[i * (width + 6) + j];
        }
    }
    return sum / (height * width);
}
__global__ void subtractMean_(float *image, int num_elements, float mean)
{

    size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);
    for (size_t i = thread_grid_idx; i < num_elements; i += num_threads_in_grid)
    {
        image[i] -= mean;
        // atomicAdd(&image[i], -mean);
    }
}
void subtractMean(float *image, int height, int width, int cub_temp_size)
{

    // hipEvent_t start, stop;
    // float time_ms;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // hipEventRecord(start);
    float mean_cuda = getMean(image, height, width, cub_temp_size);
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&time_ms, start, stop);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
                     // maximum occupancy for a full device launch
    int gridSize;    // The actual grid size needed, based on input size

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                       subtractMean_, 0, 0);
    // Round up according to array size
    gridSize = ((height) * (width) + blockSize - 1) / blockSize;

    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // hipEventRecord(start);
    subtractMean_<<<gridSize, blockSize>>>(image, height * width, mean_cuda);
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&time_ms, start, stop);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);
    // std::cout<<"actual subtractmean time: "<<time_ms<<std::endl;
}
__global__ void getContrast_(float *image, int num_elements, float *image_out)
{
    // size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    // size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);
    // for (size_t i = thread_grid_idx; i < num_elements; i += num_threads_in_grid)
    // {
    //     image_out[i] = image[i] * image[i];
    // }


    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    
    image_out[idx] = image[idx] * image[idx];
    // if (idx < num_elements)
    // {
    //     image_out[idx] = image[idx] * image[idx];
    // }
}
float getContrast(float *image, int height, int width, int cub_temp_size)
{
    float contrast = 0;

    float *temp_image;
    checkCudaErrors(hipMalloc((void **)&temp_image, (unsigned int)sizeof(float) * (height) * (width)));
    // const int num_sm = 8; // Jetson Orin NX
    // const int blocks_per_sm = 4;
    // const int threads_per_block = 128;
    int blockSize=57;   // The launch configurator returned block size
    // int minGridSize; // The minimum grid size needed to achieve the
    //                  // maximum occupancy for a full device launch
    int gridSize=768;    // The actual grid size needed, based on input size

    // hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
    //                                    getContrast_, 0, 0);
    // Round up according to array size
    // gridSize = (height * width + blockSize - 1) / blockSize;
    getContrast_<<<gridSize, blockSize>>>(image, (height) * (width), temp_image);
    // std::cout<<"getContrast "<< gridSize<<" "<< blockSize<<std::endl;
    contrast = getMean(temp_image, height, width, cub_temp_size);
    checkCudaErrors(hipFree(temp_image));
    return contrast;
}

__global__ void getContrastDel_(float *image, float *image_del, int num_elements)
{
    // size_t thread_grid_idx = size_t(blockIdx.x * blockDim.x + threadIdx.x);
    // size_t num_threads_in_grid = size_t(blockDim.x * gridDim.x);
    // for (size_t i = thread_grid_idx; i <  num_elements; i += num_threads_in_grid)
    // {
    //     image_del[i] *= image[i];
    // }
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    image_del[idx] *= image[idx];
    // if (idx < num_elements)
    // {
    //     image_del[idx] *= image[idx];
    // }
}
float getContrastDel(float *image, float *image_del, int height, int width, int cub_temp_size)
{
    float contrast = 0;
    // const int num_sm = 8; // Jetson Orin NX
    // const int blocks_per_sm = 4;
    // const int threads_per_block = 128;
    int blockSize=57;   // The launch configurator returned block size
    // int minGridSize; // The minimum grid size needed to achieve the
    //                  // maximum occupancy for a full device launch
    int gridSize=768;    // The actual grid size needed, based on input size

    // hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
    //                                    getContrastDel_, 0, 0);
    // Round up according to array size
    // gridSize = (height * width + blockSize - 1) / blockSize;
    // std::cout<<"getContrastDel "<< gridSize<<" "<< blockSize<<std::endl;
    getContrastDel_<<<gridSize, blockSize>>>(image, image_del, height* width);
    contrast = 2 * getMean(image_del, height, width, cub_temp_size);
    return contrast;
}